#include "hip/hip_runtime.h"
__constant__ k[29]

__global__ void reaction(double *species, int n)
{
  int tid = threadIdx.x;
  __shared__ s[928];
  s[tid] = species[tid];
  for (int t = 0; t < n; t++){
    if (tid < 32)
      s[tid] = -(k[0] + k[1] + k[2] + k[3] + k[4])*s[tid]*s[tid-896] + k[15]* + k[16]* + k[17]*
    else if (tid < 64)
    else if (tid < 96)
    else if (tid < 128)
    else if (tid < 160)
    else if (tid < 192)
    else if (tid < 224)
    else if (tid < 256)
    else if (tid < 288)
    else if (tid < 320)
    else if (tid < 352)
    else if (tid < 384)
    else if (tid < 416)
    else if (tid < 448)
    else if (tid < 480)
    else if (tid < 512)
    else if (tid < 544)
    else if (tid < 576)
    else if (tid < 608)
    else if (tid < 640)
    else if (tid < 672)
    else if (tid < 704)
    else if (tid < 736)
    else if (tid < 768)
    else if (tid < 800)
    else if (tid < 832)
    else if (tid < 864)
    else if (tid < 896)
    else if (tid < 928)
    __syncthreads();
  }
  species[tid] = s[tid];
}
